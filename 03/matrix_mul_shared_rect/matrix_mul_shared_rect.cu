#include "hip/hip_runtime.h"
#define PPROG_IMPLEMENTATION

#include <random.cuh>
#include <timer.cuh>
#include <utils.cuh>

// ===================================================================================
// CONFIGURATION

// A: KxN, B: NxM, C: KxM 
#define K 2
#define N 4
#define M 2

#define TILE_SIZE 32

// ===================================================================================
// UTILITIES

#define cuda_check_error()                                                     \
  {                                                                            \
    hipError_t e = hipGetLastError();                                        \
    if (e != hipSuccess) {                                                    \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__,                 \
             hipGetErrorString(e));                                           \
      std::exit(EXIT_FAILURE);                                                 \
    }                                                                          \
  }

// NOTE: use this function to visualize the matrices
void print_matrix(const std::vector<int>& matrix, int h, int w) {
    printf("\n");
    for (int y = 0; y < h; y++) {
        for (int x = 0; x < w; x++) {
            printf("%6d ", matrix[y * w + x]);
        }
        printf("\n");
    }
}

// ===================================================================================
// @@ IMPLEMENTATION

//@@ Implement the naive matrix multiplication
// A: k x n, B: n x m, C: k x m 
__global__ void matmul_gpu_naive(int *A, int* B, int *O, int k, int n, int m) {
    // @@ ...
}

//@@ Implement the tiled matrix multiplication using the shared memory
// A: k x n, B: n x m, C: k x m
__global__ void matmul_gpu_shmem(int *A, int* B, int *O, int k, int n, int m) {
    // @@ ...
}

// ===================================================================================

int main(int argc, char** argv) {

    // Generates random matrix data
    printf("generating random data...\n");
    auto A = random_matrix<int>(K, N, 2000 /* RNG SEED */);
    auto B = random_matrix<int>(N, M, 2001 /* RNG SEED */);
    printf("done\n");

    print_matrix(A, K, N);
    print_matrix(B, N, M);

    std::vector<int> result_cpu{};
    result_cpu.resize(K * M);

    std::vector<int> result_gpu_naive{};
    result_gpu_naive.resize(K * M);

    std::vector<int> result_gpu_shmem{};
    result_gpu_shmem.resize(K * M);

    // Setup blocks and threads count
    dim3 threads = dim3(TILE_SIZE, TILE_SIZE);
    dim3 blocks = dim3(
        (K + TILE_SIZE - 1) / TILE_SIZE,
        (M + TILE_SIZE - 1) / TILE_SIZE
    );

    // Allocate all necessary matrices in GPU memory
    int *d_A, *d_B, *d_O;
    hipMalloc((void**)&d_A, K * N * sizeof(int));
    hipMalloc((void**)&d_B, N * M * sizeof(int));
    hipMalloc((void**)&d_O, K * M * sizeof(int));
    cuda_check_error();

    // Copy data from CPU to GPU
    hipMemcpy(d_A, A.data(), K * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), N * M * sizeof(int), hipMemcpyHostToDevice);
    cuda_check_error();

    // ================================================================================================
    // CPU

    auto timer_cpu = timerCPU{};
    timer_cpu.start();

    result_cpu = matmul_cpu<int>(A, B, K, N, M);
    
    timer_cpu.stop();
    
    print_matrix(result_cpu, K, M);

    // ================================================================================================
    // GPU NAIVE

    auto timer_gpu_naive = timerGPU{};
    timer_gpu_naive.start();

    matmul_gpu_naive<<<blocks, threads>>>(d_A, d_B, d_O, K, N, M);

    timer_gpu_naive.stop();
    cuda_check_error();

    // Move result matrix to CPU memory
    hipMemcpy(result_gpu_naive.data(), d_O, result_gpu_naive.size() * sizeof(int), hipMemcpyDeviceToHost);
    cuda_check_error();

    //print_matrix(result_gpu_naive);

    // ================================================================================================
    // GPU SHARED MEMORY

    auto timer_gpu_shmem = timerGPU{};
    timer_gpu_shmem.start();

    // Invoke kernel
    matmul_gpu_shmem<<<blocks, threads>>>(d_A, d_B, d_O, K, N, M);
    
    timer_gpu_shmem.stop();

    // Move result matrix to CPU memory
    hipMemcpy(result_gpu_shmem.data(), d_O, result_gpu_shmem.size() * sizeof(int), hipMemcpyDeviceToHost);
    cuda_check_error();

    //print_matrix(result_gpu_shmem);

    // ================================================================================================
    // TIMERS

    auto cpu_ms = timer_cpu.elapsed_ms();
    auto gpu_naive_ms = timer_gpu_naive.elapsed_ms();
    auto gpu_shmem_ms = timer_gpu_shmem.elapsed_ms();

    printf("transpose CPU\n\t%f ms\n", cpu_ms);

    printf("transpose GPU:NAIVE\n\t%f ms (speedup: cpu %.2fx)\n", 
        gpu_naive_ms, cpu_ms / gpu_naive_ms);

    printf("transpose GPU:SHMEM\n\t%f ms (speedup: cpu %.2fx, gpu_naive %.2fx)\n", 
        gpu_shmem_ms, cpu_ms / gpu_shmem_ms, gpu_naive_ms / gpu_shmem_ms);

    // ================================================================================================
    // CHECK

    bool ok_naive = mat_check_result(result_cpu, result_gpu_naive, N);
    printf("Solution CPU vs NAIVE: %s\n", ok_naive ? "CORRECT" : "INCORRECT");

    bool ok_shmem = mat_check_result(result_cpu, result_gpu_shmem, N);
    printf("Solution CPU vs SHMEM: %s\n", ok_shmem ? "CORRECT" : "INCORRECT");

    // Free cuda memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_O);

    return 0;
}
