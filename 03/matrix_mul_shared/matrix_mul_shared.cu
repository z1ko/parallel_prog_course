#include "hip/hip_runtime.h"
#define PPROG_IMPLEMENTATION

#include <random.cuh>
#include <timer.cuh>
#include <utils.cuh>

// ===================================================================================
// CONFIGURATION

#define N 1024
#define TILE_SIZE 32

// ===================================================================================
// UTILITIES

#define cuda_check_error()                                                     \
  {                                                                            \
    hipError_t e = hipGetLastError();                                        \
    if (e != hipSuccess) {                                                    \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__,                 \
             hipGetErrorString(e));                                           \
      std::exit(EXIT_FAILURE);                                                 \
    }                                                                          \
  }

// NOTE: use this function to visualize the matrices
void print_matrix(const std::vector<int>& matrix) {
    printf("\n");
    for (int y = 0; y < N; y++) {
        for (int x = 0; x < N; x++) {
            printf("%6d ", matrix[y * N + x]);
        }
        printf("\n");
    }
}

// ===================================================================================
// @@ IMPLEMENTATION

//@@ Implement the naive matrix multiplication
__global__ void matmul_gpu_naive(int *A, int* B, int *O, int n) {
    // @@ ...
}

//@@ Implement the tiled matrix multiplication using the shared memory
__global__ void matmul_gpu_shmem(int *A, int* B, int *O, int n) {
    // @@ ...
}

// ===================================================================================

int main(int argc, char** argv) {

    // Generates random matrix data
    printf("generating random data...\n");
    auto A = random_matrix<int>(N, N, 1999 /* RNG SEED */);
    auto B = random_matrix<int>(N, N, 2000 /* RNG SEED */);
    printf("done\n");

    //print_matrix(A);
    //print_matrix(B);

    std::vector<int> result_cpu{};
    result_cpu.resize(A.size());

    std::vector<int> result_gpu_shmem{};
    result_gpu_shmem.resize(A.size());

    std::vector<int> result_gpu_naive{};
    result_gpu_naive.resize(A.size());

    // Setup blocks and threads count
    dim3 threads = dim3(TILE_SIZE, TILE_SIZE);
    dim3 blocks = dim3(
        ((N) + TILE_SIZE - 1) / TILE_SIZE,
        ((N) + TILE_SIZE - 1) / TILE_SIZE
    );

    // Allocate all necessary matrices in GPU memory
    int *d_A, *d_B, *d_O;
    hipMalloc((void**)&d_A, A.size() * sizeof(int));
    hipMalloc((void**)&d_B, A.size() * sizeof(int));
    hipMalloc((void**)&d_O, A.size() * sizeof(int));
    cuda_check_error();

    // Copy data from CPU to GPU
    hipMemcpy(d_A, A.data(), A.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), B.size() * sizeof(int), hipMemcpyHostToDevice);
    cuda_check_error();

    // ================================================================================================
    // CPU

    auto timer_cpu = timerCPU{};
    timer_cpu.start();

    result_cpu = matmul_cpu<int>(A, B, N, N, N);
    
    timer_cpu.stop();
    
    //print_matrix(result_cpu);

    // ================================================================================================
    // GPU NAIVE

    auto timer_gpu_naive = timerGPU{};
    timer_gpu_naive.start();

    matmul_gpu_naive<<<blocks, threads>>>(d_A, d_B, d_O, N);

    timer_gpu_naive.stop();
    cuda_check_error();

    // Move result matrix to CPU memory
    hipMemcpy(result_gpu_naive.data(), d_O, result_gpu_naive.size() * sizeof(int), hipMemcpyDeviceToHost);
    cuda_check_error();

    //print_matrix(result_gpu_naive);

    // ================================================================================================
    // GPU SHARED MEMORY

    auto timer_gpu_shmem = timerGPU{};
    timer_gpu_shmem.start();

    // Invoke kernel
    matmul_gpu_shmem<<<blocks, threads>>>(d_A, d_B, d_O, N);
    
    timer_gpu_shmem.stop();

    // Move result matrix to CPU memory
    hipMemcpy(result_gpu_shmem.data(), d_O, result_gpu_shmem.size() * sizeof(int), hipMemcpyDeviceToHost);
    cuda_check_error();

    //print_matrix(result_gpu_shmem);

    // ================================================================================================
    // TIMERS

    auto cpu_ms = timer_cpu.elapsed_ms();
    auto gpu_naive_ms = timer_gpu_naive.elapsed_ms();
    auto gpu_shmem_ms = timer_gpu_shmem.elapsed_ms();

    printf("transpose CPU\n\t%f ms\n", cpu_ms);

    printf("transpose GPU:NAIVE\n\t%f ms (speedup: cpu %.2fx)\n", 
        gpu_naive_ms, cpu_ms / gpu_naive_ms);

    printf("transpose GPU:SHMEM\n\t%f ms (speedup: cpu %.2fx, gpu_naive %.2fx)\n", 
        gpu_shmem_ms, cpu_ms / gpu_shmem_ms, gpu_naive_ms / gpu_shmem_ms);

    // ================================================================================================
    // CHECK

    bool ok_naive = mat_check_result(result_cpu, result_gpu_naive, N);
    printf("Solution CPU vs NAIVE: %s\n", ok_naive ? "CORRECT" : "INCORRECT");

    bool ok_shmem = mat_check_result(result_cpu, result_gpu_shmem, N);
    printf("Solution CPU vs SHMEM: %s\n", ok_shmem ? "CORRECT" : "INCORRECT");

    // Free cuda memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_O);

    return 0;
}
